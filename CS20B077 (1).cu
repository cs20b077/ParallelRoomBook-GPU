
#include <iostream>
#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>

#define max_N 100000
#define max_P 30
#define BLOCKSIZE 1024

using namespace std;

typedef struct request  //struct for storing requests
{
    int id;
    int centre;
    int facility;
    int newfacility;
    int start;
    int slots;
}req;

//*******************************************

// Write down the kernels here
__global__ void calprefix(req *reqs, int *presum, int R)  //kernel for calculating prefix sum
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int i = tid;
    if(i<R)
    {
        atomicAdd(&presum[reqs[i].newfacility],1);        
    }
}

__global__ void final(int *presum, req *reqs, int *success, int *succreqs, int N,int *capacity) //kernel for final allocation(finding success and failures)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;            //thread id(facility number)
    int k = tid;
    if(k<N)
    {
        int len;                                                //length of requests for a particular facility
        if(k==0){
            len = presum[k];                                    //if facility number is 0 then length is equal to prefix sum of facility number 0   
        }
        else{
            len = presum[k] - presum[k-1];                      //else length is equal to prefix sum of facility number k - prefix sum of facility number k-1
        }
        if(len!=0){
          int l=presum[k]-len;                                  //left index of requests for a particular facility                  
          int r=presum[k]-1;                                    //right index of requests for a particular facility
          int cap=capacity[reqs[l].newfacility];                //capacity of facility
          int slots[25];                                        //array to store slots of facility      
          for(int i=0;i<25;i++){
            slots[i]=cap;
          }
          /*printf("id-%d before ",k);
          for(int i=0;i<25;i++){
            printf("%d ",slots[i]);
          }*/
          int i;
          for(i=l;i<=r;i++){
            int flag=0;
            for(int j=reqs[i].start;j<reqs[i].start+reqs[i].slots;j++){  //checking if slots are available
              if(slots[j]==0){
                flag=1;
                break;
              }
            }
            //printf("hellob-%d",k);
            if(flag==0){
              for(int j=reqs[i].start;j<reqs[i].start+reqs[i].slots;j++){
                slots[j]--;
                //facilityslots[reqs[i].newfacility*max_P+j]+=1;
              }
              //printf("helloa-%d %d\n",k,success[0]);
              atomicAdd(&success[0],1);                                 //incrementing success
              atomicAdd(&succreqs[reqs[i].centre],1);                   //incrementing success for a particular centre
            }
          }
          /*printf("after");
          for(int i=0;i<25;i++){
            printf("%d ",slots[i]);
          }
          printf("\n");*/
        }
    }
}
//***********************************************

//comparator funtion using both newfacility and id
int compare(const void *a, const void *b)                               //comparator function for sorting requests
{
    req *x = (req *)a;
    req *y = (req *)b;
    if(x->newfacility == y->newfacility)                                //if newfacility is same then sort according to id
    {
        return x->id - y->id;
    }
    return x->newfacility - y->newfacility;                             //else sort according to newfacility
}

int main(int argc,char **argv)
{
	// variable declarations...
    int N,*centre,*facility,*capacity,*fac_ids, *succ_reqs, *tot_reqs;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	
    // Allocate memory on cpu
    centre=(int*)malloc(N * sizeof (int));  // Computer  centre numbers
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer centre
    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer centre
    capacity=(int*)malloc(max_P * N * sizeof (int));  // stores capacities of each facility for every computer centre 


    int success=0;  // total successful requests
    int fail = 0;   // total failed requests
    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each centre
    succ_reqs = (int *)malloc(N*sizeof(int)); // total successful requests for each centre

    // Input the computer centres data
    int k1=0 , k2 = 0;
    for(int i=0;i<N;i++)
    {
      fscanf( inputfilepointer, "%d", &centre[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[i*max_P+j] );
        k1++;
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[i*max_P+j]);
        k2++;     
      }
    }

    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots;   // Number of slots requested for every request
    
    // Allocate memory on CPU 
	int R;
	fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ( (R) * sizeof (int) );  // Request ids
    req_cen = (int *) malloc ( (R) * sizeof (int) );  // Requested computer centre
    req_fac = (int *) malloc ( (R) * sizeof (int) );  // Requested facility
    req_start = (int *) malloc ( (R) * sizeof (int) );  // Start slot of every request
    req_slots = (int *) malloc ( (R) * sizeof (int) );   // Number of slots requested for every request
    

    //****** by me
    // struct 
    req *reqs = (req *)malloc(R * sizeof(req));         // array of requests
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       reqs[j].id = req_id[j];
       reqs[j].centre = req_cen[j];
       reqs[j].facility = req_fac[j];
       reqs[j].newfacility = req_cen[j]*max_P+req_fac[j]; // new facility id(current_centre*max_P+facility)
       reqs[j].start = req_start[j];
       reqs[j].slots = req_slots[j];  
       tot_reqs[req_cen[j]]+=1;  
    }
    // sort the requests
    qsort(reqs, R, sizeof(req), compare);                 //sorting requests according to newfacility and id
    /*for(int i=0;i<R;i++){
      printf("%d %d %d %d %d %d\n",reqs[i].id,reqs[i].centre,reqs[i].facility,reqs[i].newfacility,reqs[i].start,reqs[i].slots);
    }
    */
    //*********************************
    //*********************************
    // Call the kernels here
    // Allocate memory on GPU
    int *d_capacity;                                    // stores capacities of each facility for every computer centre
    hipMalloc(&d_capacity, N*max_P*sizeof(int));
    hipMemcpy(d_capacity, capacity, N*max_P*sizeof(int), hipMemcpyHostToDevice);
    int *d_success;                                     // total successful requests
    hipMalloc(&d_success, sizeof(int));
    hipMemset(d_success, 0, sizeof(int));
    int *d_succreqs;                                    // total successful requests for each centre
    hipMalloc(&d_succreqs, N*sizeof(int));
    hipMemset(d_succreqs, 0, N*sizeof(int));
    int *presum=(int*)malloc(N *max_P* sizeof (int));   // prefix sum of requests for each facility
    int *d_presum;
    hipMalloc(&d_presum, N*max_P*sizeof(int));
    hipMemset(d_presum, 0, N*max_P*sizeof(int));
    req *d_reqs;
    hipMalloc(&d_reqs, R*sizeof(req));
    hipMemcpy(d_reqs, reqs, R*sizeof(req), hipMemcpyHostToDevice);
    dim3 grid1((R+1023)/1024,1,1);                                          //grid and block size for kernel call
    dim3 block1(1024,1,1);
    calprefix<<<grid1,block1>>>(d_reqs, d_presum, R);     //calculating request for each facility
    hipMemcpy(presum, d_presum, N*max_P*sizeof(int), hipMemcpyDeviceToHost);
    int x=0;
    for(int i=0;i<N*max_P;i++){
        if(i!=0){
            presum[i]+=presum[i-1];                     //calculating prefix sum
        }
    }
    int *d_presum1;
    hipMalloc(&d_presum1, N*max_P*sizeof(int));
    hipMemcpy(d_presum1, presum, N*max_P*sizeof(int), hipMemcpyHostToDevice);
    //remove again
    /*int *facilityslots;
    facilityslots=(int*)malloc((N*max_P*24+1)*sizeof(int));
    */
    //rem
    //int *d_facilityslots;
    /*cudaMalloc(&d_facilityslots, (N*max_P*24+1)*sizeof(int));
    cudaMemset(d_facilityslots, 0, (N*max_P*24+1)*sizeof(int));
    */
    dim3 grid2((N*max_P+1023)/1024,1,1);                                          //grid and block size for kernel call
    dim3 block2(1024,1,1);
    final<<<grid2,block2>>>(d_presum1,d_reqs,d_success,d_succreqs,N*max_P,d_capacity);  //final kernel call(success calculation) parallelism on facilities
    //cudaMemcpy(facilityslots, d_facilityslots, (N*max_P*24+1)*sizeof(int), cudaMemcpyDeviceToHost);
    hipMemcpy(&success, d_success, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(succ_reqs, d_succreqs, N*sizeof(int), hipMemcpyDeviceToHost);
    fail=R-success;
    /*for(int i=0;i<N;i++){
      printf("center-%d\n",i);
      for(int j=0;j<max_P;j++){
        printf("fac-%d ",j);
        for(int k=1;k<=24;k++){
          printf("%d ",facilityslots[i*max_P*24+j*24+k]);
        }
        printf("\n");
      }
    }
    */
    //********************************
    /*printf("success-%d",success);
    for(int i=0;i<N;i++){
      printf("successcenter%d-%d",i,succ_reqs[i]);
    }
    */

    // Output
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    fprintf( outputfilepointer, "%d %d\n", success, fail);
    //printf("**********************************\n");
    //printf("%d %d\n", success, fail);
    for(int j = 0; j < N; j++)
    {
        fprintf( outputfilepointer, "%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
        //printf("%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
    }
    /*for(int i=0;i<N;i++){
      fprintf( outputfilepointer, "presum-Center %d\n",i);
      for(int j=0;j<max_P;j++){
        fprintf( outputfilepointer, "%d ",presum[i*max_P+j]);
      }
      fprintf( outputfilepointer, "\n");
    }
    */
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	return 0;
}